#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <time.h>

#define NUM_BLOCKS 65535

using namespace std;

__device__ __host__
int mIndex(int i, int j, int n, int m) {
    return i*m + j;
}

int64_t cache[100];
int64_t cached = 0;
int64_t factorial(int64_t n) {
    if (n < 2) return 1;
    if (n > cached) {
        cache[n] = n * factorial(n-1);
        cached = n;
        // cout << "Calculating factorial " << n <<": " << cache[n] << endl;
    }
    return cache[n];
}

void printOrders(int* orders, int numCustomers, int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, j, numCustomers, numProducts)] << " ";
        }
        cout << endl;
    }
}

void printSet(int set, int numProducts) {
    for (int i = 0; i < numProducts; i++) {
        cout << set % 2 << " ";
        set /= 2;
    }
    cout << endl;
}

void printOrdersInSequence(int* sequence,
                           int* orders,
                           int numCustomers,
                           int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, sequence[j], numCustomers, numProducts)]
                 << " ";
        }
        cout << endl;
    }
}

// Calculates the maximum number of open stacks for a given producing sequence
__device__ __host__
int maximumOpenStacks(int* sequence,
                      int* orders,
                      int numCustomers,
                      int numProducts) {
    int* toDo = (int*) malloc(numCustomers * sizeof(int));
    int* done = (int*) malloc(numCustomers * sizeof(int));
    for (int customer = 0; customer < numCustomers; customer++) {
        done[customer] = 0;
        toDo[customer] = 0;
        for (int product = 0; product < numProducts; product++) {
            // suppose that orders has only 1's and 0's
            toDo[customer] += orders[mIndex(customer, product, numCustomers,
                                            numProducts)];
        }
    }

    int numOpenStacks = 0;
    for (int i = 0; i < numProducts; i++) {
        int product = sequence[i];
        for (int customer = 0; customer < numCustomers; customer++) {
            if (orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                toDo[customer]--;
                done[customer]++;
            }
        }
        int currentOpenStacks = 0;
        for (int customer = 0; customer < numCustomers; customer++) {
            if ((done[customer] > 0 && toDo[customer] > 0) ||
                orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                currentOpenStacks++;
            }
        }
        if (currentOpenStacks > numOpenStacks) {
            numOpenStacks = currentOpenStacks;
        }
    }

    free(toDo);
    free(done);

    return numOpenStacks;
}

__device__ __host__
void generateSequence(int* sequence, int k, int numProducts) {
    for (int i = 0; i < numProducts; i++) {
        sequence[i] = i;
    }
    for (int i = 0; i < numProducts; i++) {
        int temp = sequence[k % (i + 1)];
        sequence[k % (i + 1)] = sequence[i];
        sequence[i] = temp;
        k = k / (i + 1);
    }
}

__global__
void calculateMaximumOpenStacks(int* stackSizes,
                                int* orders,
                                int numCustomers,
                                int numProducts,
                                int step) {
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, step * NUM_BLOCKS + blockIdx.x, numProducts);
    stackSizes[blockIdx.x] = maximumOpenStacks(sequence,
                                               orders,
                                               numCustomers,
                                               numProducts);
    free(sequence);
}

void checkOk(hipError_t err) {
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

void bruteForceSolve(int* orders,
                     int numCustomers,
                     int numProducts) {
    int* orders_d;
    int sizeOrders = numCustomers * numProducts * sizeof(int);
    checkOk(hipMalloc((void**) &orders_d, sizeOrders));
    checkOk(hipMemcpy(orders_d, orders, sizeOrders, hipMemcpyHostToDevice));

    int* stackSizes_d;
    int64_t numSequences = factorial(numProducts);
    int sizeStacksSizes = NUM_BLOCKS * sizeof(int);
    checkOk(hipMalloc((void**) &stackSizes_d, sizeStacksSizes));

    cout << "numSequences: " << numSequences << endl;

    int* stackSizes = (int*) malloc(sizeStacksSizes);
    int minStacks = numCustomers + 1;
    int bestK = -1;
    for (int i = 0; i < ceil(numSequences/NUM_BLOCKS); i++) {
        int numSequencesToProcess;
        if (numSequences - i * NUM_BLOCKS >= NUM_BLOCKS)
            numSequencesToProcess = NUM_BLOCKS;
        else
            numSequencesToProcess = numSequences - i * NUM_BLOCKS;

        cout << "Step " << i << ". Calculating " << numSequencesToProcess
             << " More " << numSequences - i * NUM_BLOCKS << " to go." << endl;

        // Calculating maximum stack for each one of them
        calculateMaximumOpenStacks<<<numSequencesToProcess, 1>>>(stackSizes_d,
                                                                 orders_d,
                                                                 numCustomers,
                                                                 numProducts,
                                                                 i);

        checkOk(hipMemcpy(stackSizes,
                           stackSizes_d,
                           sizeStacksSizes,
                           hipMemcpyDeviceToHost));

        // Calculate the global minimum
        for (int j = 0; j < numSequencesToProcess; j++) {
            if (stackSizes[j] < minStacks) {
                minStacks = stackSizes[j];
                bestK = j + i * NUM_BLOCKS;
            }
        }
    }

    checkOk(hipFree(orders_d));
    checkOk(hipFree(stackSizes_d));

    free(stackSizes);

    // Debugging output

    cout << "minStacks: " << minStacks << endl;

    // Print sequence
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, bestK, numProducts);
    cout << "Best sequence:" << endl;
    for (int i = 0; i < numProducts; i++) {
        cout << sequence[i] << " ";
    }
    cout << endl;

    // See orders being produced
    printOrdersInSequence(sequence, orders, numCustomers, numProducts);
    cout << "Open stacks: "
         << maximumOpenStacks(sequence, orders, numCustomers, numProducts)
         << endl;

    free(sequence);
    // End of debugging code
}

int64_t combination(int n, int k) {
    return factorial(n)/factorial(k)/factorial(n-k);
}

bool contains(int set, int p) {
    for (int i = 0; i < p; i++) {
        set /= 2;
    }
    // cout << set << endl;
    return set % 2;
}

int remove(int set, int p) {
    int stack = 0;
    int offset = 1;
    for (int i = 0; i < 32; i++) {
        if (i != p)
            stack += (set % 2) * offset;
        set /= 2;
        offset *= 2;
    }
    return stack;
}

int a(int p,
      int set,
      int* orders,
      int numCustomers,
      int numProducts) {
    bool* before = (bool*) malloc(numCustomers * sizeof(bool));
    bool* after = (bool*) malloc(numCustomers * sizeof(bool));
    bool* now = (bool*) malloc(numCustomers * sizeof(bool));
    for (int i = 0; i < numCustomers; i++) {
        before[i] = false;
        after[i] = false;
        now[i] = false;
    }

    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            if (j == p &&
                orders[mIndex(i, j, numCustomers, numProducts)] > 0) {
                now[i] = true;
            }
            if (contains(set, j) == true &&
                orders[mIndex(i, j, numCustomers, numProducts)] > 0) {
                after[i] = true;
            }
            if (contains(set, j) == false &&
                orders[mIndex(i, j, numCustomers, numProducts)] > 0) {
                before[i] = true;
            }
        }
    }

    // cout << "Testing after and before for set: " << set << " and product " << p << endl;
    // printSet(set, numProducts);

    int active_stacks = 0;
    for (int i = 0; i < numCustomers; i++) {
        if(now[i] || (before[i] && after[i])) {
            active_stacks++;
            // cout << i << " ";
        }
    }
    // cout << endl;

    free(now);
    free(after);
    free(before);

    // cout << active_stacks << " active" << endl;

    return active_stacks;
}

void computeStacks(int set,
                   int* stacksResults,
                   int* bestP,
                   int* orders,
                   int numCustomers,
                   int numProducts) {
    // cout << "Computing stacks for set: " << set << endl;
    // printSet(set, numProducts);

    if (set == 0) {
        stacksResults[set] = 0;
        return;
    }

    // cout << endl;
    int best = -1;
    int min_stacks = numCustomers;
    for (int p = 0; p < numProducts; p++) {
        if (contains(set, p)) {
            int newSet = remove(set, p);
            // cout << "Using result for set: " << newSet << " which is " << stacksResults[newSet] << endl;
            // printSet(newSet, numProducts);
            int active = a(p, newSet, orders, numCustomers, numProducts);
            int after = stacksResults[newSet];
            int max = (active > after) ? active : after;
            if (max < min_stacks) {
                min_stacks = max;
                best = p;
            }
        }
    }
    // cout << endl;
    stacksResults[set] = min_stacks;
    bestP[set] = best;
}

int countOnes(int n) {
    int ones = 0;
    while (n > 0) {
        ones += n % 2;
        n /= 2;
    }
    return ones;
}

void dpSolve(int* orders, int numCustomers, int numProducts) {
    int** sets = (int**) malloc((numProducts + 1) * sizeof(int*));
    int* combinations = (int*) malloc((numProducts + 1) * sizeof(int*));
    for (int i = 0; i < (numProducts + 1); i++) {
        int numCombinations = combination(numProducts, i);
        sets[i] = (int*) malloc(numCombinations * sizeof(int));
        combinations[i] = 0;
    }

    for (int i = 0; i < pow(2, numProducts); i++) {
        int ones = countOnes(i);
        sets[ones][combinations[ones]] = i;
        combinations[ones]++;
    }

    int* stacksResults = (int*) malloc(pow(2, numProducts) * sizeof(int));
    int* bestP = (int*) malloc(pow(2, numProducts) * sizeof(int));
    for (int setSize = 0;  setSize < numProducts + 1;  setSize++) {
        for (int setIndex = 0; setIndex < combinations[setSize]; setIndex++) {
            computeStacks(sets[setSize][setIndex],
                          stacksResults,
                          bestP,
                          orders,
                          numCustomers,
                          numProducts);
        }
    }


    // cout << "bestP" << endl;
    // for (int i = 0; i < pow(2, numProducts); i++) {
    //     cout << bestP[i] << " ";
    // }
    // cout << endl;

    // cout << "stacksResults" << endl;
    // for (int i = 0; i < pow(2, numProducts); i++) {
    //     cout << stacksResults[i] << " ";
    // }
    // cout << endl;

    cout << "Best sequence:" << endl;
    int set = pow(2, numProducts) - 1;
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    for (int i = 0; i < numProducts; i++) {
        int best = bestP[set];
        set = remove(set, best);
        sequence[i] = best;
        cout << best << " ";
    }
    cout << endl;
    printOrdersInSequence(sequence, orders, numCustomers, numProducts);

    cout << "OpenStacks: " << stacksResults[(int) pow(2, numProducts) - 1] << endl;

    // Freeing memory
}

int main(int argc, char** argv) {
    ifstream readFile;
    int numCustomers, numProducts;
    int* orders;

    char* input;
    if (argc > 1) {
        input = argv[1];
    } else {
        input = "input.txt";
    }
    cout << "Reading from " << input << endl;
    readFile.open(input);

    if (readFile.is_open()) {
        readFile >> numCustomers;
        readFile >> numProducts;
        orders = (int*) malloc(numCustomers * numProducts * sizeof(int));
        for (int i = 0; i < numCustomers; i++) {
            for (int j = 0; j < numProducts; j++) {
                int output;
                readFile >> output;
                orders[mIndex(i, j, numCustomers, numProducts)] = output;
            }
        }
        readFile.close();
    } else {
        cout << "Not able to open the input file." <<  endl;
    }
    cout << "numCustomers: " << numCustomers << endl
         << "numProducts: " << numProducts << endl;
    printOrders(orders, numCustomers, numProducts);
    clock_t start = clock();
    // bruteForceSolve(orders, numCustomers, numProducts);
    dpSolve(orders, numCustomers, numProducts);
    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    cout << "Took " << seconds << " seconds" << endl;
    return 0;
}
