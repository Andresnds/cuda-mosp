#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>

using namespace std;

__device__ __host__
int mIndex(int i, int j, int n, int m) {
  return i*m + j;
}

int factorial(int n) {
    return tgamma(n + 1);
}

// Calculates the maximum number of open stacks for a given producing sequence
__device__
int maximumOpenStacks(int* sequence,
                      int* orders,
                      int numCustomers,
                      int numProducts) {
    int* toDo = (int*) malloc(numCustomers * sizeof(int));
    int* done = (int*) malloc(numCustomers * sizeof(int));
    for (int customer = 0; customer < numCustomers; customer++) {
        done[customer] = 0;
        for (int product = 0; product < numProducts; product++) {
            // suppose that orders has only 1's and 0's
            toDo[customer] += orders[mIndex(customer, product, numCustomers,
                                            numProducts)];
        }
    }

    int numOpenStacks = 0;
    for (int i = 0; i < numProducts; i++) {
        int product = sequence[i];
        for (int customer = 0; customer < numCustomers; customer++) {
            toDo[customer] -= orders[mIndex(customer, product, numCustomers,
                                            numProducts)];
            done[customer] += orders[mIndex(customer, product, numCustomers,
                                            numProducts)];
        }
        int currentOpenStacks = 0;
        for (int customer = 0; customer < numCustomers; customer++) {
            if ((done[customer] > 0 && toDo[customer] > 0) ||
                orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                currentOpenStacks++;
            }
        }
        if (currentOpenStacks > numOpenStacks) {
            numOpenStacks = currentOpenStacks;
        }
    }

    free(toDo);
    free(done);

    return numOpenStacks;
}

__global__
void generateSequence(int* sequences, int numProducts) {
    int begin = blockIdx.x * numProducts;
    int end = begin + numProducts;
    int k = blockIdx.x;
    for (int i = begin; i < end; i++) {
        sequences[i] = i;
    }
    for (int i = begin; i < end; i++) {
        int temp = sequences[k % (i + 1)];
        sequences[k % (i + 1)] = sequences[i];
        sequences[i] = temp;
        k = k / (i + 1);
    }
}

__global__
void calculateMaximumOpenStacks(int* sequences,
                                int* stackSizes,
                                int* orders,
                                int numCustomers,
                                int numProducts) {
    int begin = blockIdx.x * numProducts;
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    for (int i = 0; i < numProducts; i++) {
        sequence[i] = sequences[begin + i];
    }

    stackSizes[blockIdx.x] = maximumOpenStacks(sequence,
                                               orders,
                                               numCustomers,
                                               numProducts);
    free(sequence);
}

void bruteForceSolve(int* orders,
                     int numCustomers,
                     int numProducts) {

    int* orders_d;
    int sizeOrders = numCustomers * numProducts * sizeof(int);
    hipMalloc((void**) &orders_d, sizeOrders);
    hipMemcpy(orders_d, orders, sizeOrders, hipMemcpyHostToDevice);

    int* sequences_d;
    int numSequences = factorial(numProducts);
    int sizeSequences = numSequences * numProducts * sizeof(int);
    hipMalloc((void**) &sequences_d, sizeSequences);

    // Generating all possible sequences
    generateSequence<<<numSequences, 1>>>(sequences_d, numProducts);

    int* stackSizes_d;
    int sizeStacksSizes = numSequences * sizeof(int);
    hipMalloc((void**) &stackSizes_d, sizeStacksSizes);

    // Calculating maximum stack for each one of them
    calculateMaximumOpenStacks<<<numSequences, 1>>>(sequences_d,
                                                    stackSizes_d,
                                                    orders_d,
                                                    numCustomers,
                                                    numProducts);

    int* stackSizes = (int*) malloc(sizeStacksSizes);
    hipMemcpy(stackSizes,
               stackSizes_d,
               sizeStacksSizes,
               hipMemcpyDeviceToHost);

    hipFree(orders_d);
    hipFree(sequences_d);
    hipFree(stackSizes_d);

    // Calculate the global minimum
    int minStacks = numCustomers + 1;
    for (int i = 0; i < numSequences; i++) {
        if (stackSizes[i] < minStacks) {
            minStacks = stackSizes[i];
        }
    }
    cout << "minStacks: " << minStacks << endl;

    free(stackSizes);
}

void dpSolve(int* orders) {
    // for each size of set generate the number of
}

void printOrders(int* orders, int numCustomers, int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, j, numCustomers, numProducts)] << " ";
        }
        cout << endl;
    }
}

int main(int argc, char** argv) {
    ifstream readFile;
    int numCustomers, numProducts;
    int* orders;

    string input;
    if (argc > 1) {
        input = argv[1];
    } else {
        input = "input.txt";
    }
    cout << "Reading from " << input << endl;
    readFile.open(input);

    if (readFile.is_open()) {
        readFile >> numCustomers;
        readFile >> numProducts;
        orders = (int*) malloc(numCustomers * numProducts * sizeof(int));
        for (int i = 0; i < numCustomers; i++) {
            for (int j = 0; j < numProducts; j++) {
                int output;
                readFile >> output;
                orders[mIndex(i, j, numCustomers, numProducts)] = output;
            }
        }
        readFile.close();
    } else {
        cout << "Not able to open the input file." <<  endl;
    }
    cout << "numCustomers: " << numCustomers << endl
         << "numProducts: " << numProducts << endl;
    printOrders(orders, numCustomers, numProducts);
    bruteForceSolve(orders, numCustomers, numProducts);
    return 0;
}