#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>

using namespace std;

__device__ __host__
int mIndex(int i, int j, int n, int m) {
  return i*m + j;
}

int factorial(int n) {
    return tgamma(n + 1);
}

void printOrders(int* orders, int numCustomers, int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, j, numCustomers, numProducts)] << " ";
        }
        cout << endl;
    }
}

void printOrdersInSequence(int* sequence,
                           int* orders,
                           int numCustomers,
                           int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, sequence[j], numCustomers, numProducts)]
                 << " ";
        }
        cout << endl;
    }
}

// Calculates the maximum number of open stacks for a given producing sequence
__device__ __host__
int maximumOpenStacks(int* sequence,
                      int* orders,
                      int numCustomers,
                      int numProducts) {
    int* toDo = (int*) malloc(numCustomers * sizeof(int));
    int* done = (int*) malloc(numCustomers * sizeof(int));
    for (int customer = 0; customer < numCustomers; customer++) {
        done[customer] = 0;
        toDo[customer] = 0;
        for (int product = 0; product < numProducts; product++) {
            // suppose that orders has only 1's and 0's
            toDo[customer] += orders[mIndex(customer, product, numCustomers,
                                            numProducts)];
        }
    }

    int numOpenStacks = 0;
    for (int i = 0; i < numProducts; i++) {
        int product = sequence[i];
        for (int customer = 0; customer < numCustomers; customer++) {
            if (orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                toDo[customer]--;
                done[customer]++;
            }
        }
        int currentOpenStacks = 0;
        for (int customer = 0; customer < numCustomers; customer++) {
            if ((done[customer] > 0 && toDo[customer] > 0) ||
                orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                currentOpenStacks++;
            }
        }
        if (currentOpenStacks > numOpenStacks) {
            numOpenStacks = currentOpenStacks;
        }
    }

    free(toDo);
    free(done);

    return numOpenStacks;
}

__device__ __host__
void generateSequence(int* sequence, int k, int numProducts) {
    for (int i = 0; i < numProducts; i++) {
        sequence[i] = i;
    }
    for (int i = 0; i < numProducts; i++) {
        int temp = sequence[k % (i + 1)];
        sequence[k % (i + 1)] = sequence[i];
        sequence[i] = temp;
        k = k / (i + 1);
    }
}

__global__
void calculateMaximumOpenStacks(int* stackSizes,
                                int* orders,
                                int numCustomers,
                                int numProducts) {
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, blockIdx.x, numProducts);
    stackSizes[blockIdx.x] = maximumOpenStacks(sequence,
                                               orders,
                                               numCustomers,
                                               numProducts);
    free(sequence);
}

void checkOk(hipError_t err) {
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

void bruteForceSolve(int* orders,
                     int numCustomers,
                     int numProducts) {
    int* orders_d;
    int sizeOrders = numCustomers * numProducts * sizeof(int);
    checkOk(hipMalloc((void**) &orders_d, sizeOrders));
    checkOk(hipMemcpy(orders_d, orders, sizeOrders, hipMemcpyHostToDevice));

    int* stackSizes_d;
    int numSequences = factorial(numProducts);
    int sizeStacksSizes = numSequences * sizeof(int);
    checkOk(hipMalloc((void**) &stackSizes_d, sizeStacksSizes));

    // Calculating maximum stack for each one of them
    calculateMaximumOpenStacks<<<numSequences, 1>>>(stackSizes_d,
                                                    orders_d,
                                                    numCustomers,
                                                    numProducts);

    int* stackSizes = (int*) malloc(sizeStacksSizes);
    checkOk(hipMemcpy(stackSizes,
                       stackSizes_d,
                       sizeStacksSizes,
                       hipMemcpyDeviceToHost));

    checkOk(hipFree(orders_d));
    checkOk(hipFree(stackSizes_d));

    // Calculate the global minimum
    int minStacks = numCustomers + 1;
    int bestK = -1;
    for (int i = 0; i < numSequences; i++) {
        if (stackSizes[i] < minStacks) {
            minStacks = stackSizes[i];
            bestK = i;
        }
    }
    cout << "minStacks: " << minStacks << endl;

    // Debugging output

    // Print sequence
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, bestK, numProducts);
    cout << "Best sequence:" << endl;
    for (int i = 0; i < numProducts; i++) {
        cout << sequence[i] << " ";
    }
    cout << endl;

    // See orders being produced
    printOrdersInSequence(sequence, orders, numCustomers, numProducts);
    cout << "Open stacks: "
         << maximumOpenStacks(sequence, orders, numCustomers, numProducts)
         << endl;

    free(sequence);
    // End of debugging code


    free(stackSizes);
}

void dpSolve(int* orders) {
    // for each size of set generate the number of
}

int main(int argc, char** argv) {
    ifstream readFile;
    int numCustomers, numProducts;
    int* orders;

    string input;
    if (argc > 1) {
        input = argv[1];
    } else {
        input = "input.txt";
    }
    cout << "Reading from " << input << endl;
    readFile.open(input);

    if (readFile.is_open()) {
        readFile >> numCustomers;
        readFile >> numProducts;
        orders = (int*) malloc(numCustomers * numProducts * sizeof(int));
        for (int i = 0; i < numCustomers; i++) {
            for (int j = 0; j < numProducts; j++) {
                int output;
                readFile >> output;
                orders[mIndex(i, j, numCustomers, numProducts)] = output;
            }
        }
        readFile.close();
    } else {
        cout << "Not able to open the input file." <<  endl;
    }
    cout << "numCustomers: " << numCustomers << endl
         << "numProducts: " << numProducts << endl;
    printOrders(orders, numCustomers, numProducts);
    bruteForceSolve(orders, numCustomers, numProducts);
    return 0;
}