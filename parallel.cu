#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <time.h>

#define NUM_BLOCKS 65535

using namespace std;

__device__ __host__
int mIndex(int i, int j, int n, int m) {
    return i*m + j;
}

double factorial(int n) {
    double x = n + 1;
    return tgamma(x);
}

void printOrders(int* orders, int numCustomers, int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, j, numCustomers, numProducts)] << " ";
        }
        cout << endl;
    }
}

void printOrdersInSequence(int* sequence,
                           int* orders,
                           int numCustomers,
                           int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, sequence[j], numCustomers, numProducts)]
                 << " ";
        }
        cout << endl;
    }
}

// Calculates the maximum number of open stacks for a given producing sequence
__device__ __host__
int maximumOpenStacks(int* sequence,
                      int* orders,
                      int numCustomers,
                      int numProducts) {
    int* toDo = (int*) malloc(numCustomers * sizeof(int));
    int* done = (int*) malloc(numCustomers * sizeof(int));
    for (int customer = 0; customer < numCustomers; customer++) {
        done[customer] = 0;
        toDo[customer] = 0;
        for (int product = 0; product < numProducts; product++) {
            // suppose that orders has only 1's and 0's
            toDo[customer] += orders[mIndex(customer, product, numCustomers,
                                            numProducts)];
        }
    }

    int numOpenStacks = 0;
    for (int i = 0; i < numProducts; i++) {
        int product = sequence[i];
        for (int customer = 0; customer < numCustomers; customer++) {
            if (orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                toDo[customer]--;
                done[customer]++;
            }
        }
        int currentOpenStacks = 0;
        for (int customer = 0; customer < numCustomers; customer++) {
            if ((done[customer] > 0 && toDo[customer] > 0) ||
                orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                currentOpenStacks++;
            }
        }
        if (currentOpenStacks > numOpenStacks) {
            numOpenStacks = currentOpenStacks;
        }
    }

    free(toDo);
    free(done);

    return numOpenStacks;
}

__device__ __host__
void generateSequence(int* sequence, int k, int numProducts) {
    for (int i = 0; i < numProducts; i++) {
        sequence[i] = i;
    }
    for (int i = 0; i < numProducts; i++) {
        int temp = sequence[k % (i + 1)];
        sequence[k % (i + 1)] = sequence[i];
        sequence[i] = temp;
        k = k / (i + 1);
    }
}

__global__
void calculateMaximumOpenStacks(int* stackSizes,
                                int* orders,
                                int numCustomers,
                                int numProducts,
                                int step) {
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, step * NUM_BLOCKS + blockIdx.x, numProducts);
    stackSizes[blockIdx.x] = maximumOpenStacks(sequence,
                                               orders,
                                               numCustomers,
                                               numProducts);
    free(sequence);
}

void checkOk(hipError_t err) {
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

void bruteForceSolve(int* orders,
                     int numCustomers,
                     int numProducts) {
    int* orders_d;
    int sizeOrders = numCustomers * numProducts * sizeof(int);
    checkOk(hipMalloc((void**) &orders_d, sizeOrders));
    checkOk(hipMemcpy(orders_d, orders, sizeOrders, hipMemcpyHostToDevice));

    int* stackSizes_d;
    double numSequences = factorial(numProducts);
    int sizeStacksSizes = NUM_BLOCKS * sizeof(int);
    checkOk(hipMalloc((void**) &stackSizes_d, sizeStacksSizes));

    cout << "numSequences: " << numSequences << endl;

    int* stackSizes = (int*) malloc(sizeStacksSizes);
    int minStacks = numCustomers + 1;
    int bestK = -1;
    for (int i = 0; i < ceil(numSequences/NUM_BLOCKS); i++) {
        int numSequencesToProcess;
        if (numSequences - i * NUM_BLOCKS >= NUM_BLOCKS)
            numSequencesToProcess = NUM_BLOCKS;
        else
            numSequencesToProcess = numSequences - i * NUM_BLOCKS;

        // cout << "Step " << i << ". Calculating " << numSequencesToProcess
        //      << " More " << numSequences - i * NUM_BLOCKS << " to go." << endl;

        // Calculating maximum stack for each one of them
        calculateMaximumOpenStacks<<<numSequencesToProcess, 1>>>(stackSizes_d,
                                                                 orders_d,
                                                                 numCustomers,
                                                                 numProducts,
                                                                 i);

        checkOk(hipMemcpy(stackSizes,
                           stackSizes_d,
                           sizeStacksSizes,
                           hipMemcpyDeviceToHost));

        // Calculate the global minimum
        for (int j = 0; j < numSequencesToProcess; j++) {
            if (stackSizes[j] < minStacks) {
                minStacks = stackSizes[j];
                bestK = j + i * NUM_BLOCKS;
            }
        }
    }

    checkOk(hipFree(orders_d));
    checkOk(hipFree(stackSizes_d));

    free(stackSizes);

    // Debugging output

    cout << "minStacks: " << minStacks << endl;

    // Print sequence
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, bestK, numProducts);
    cout << "Best sequence:" << endl;
    for (int i = 0; i < numProducts; i++) {
        cout << sequence[i] << " ";
    }
    cout << endl;

    // See orders being produced
    printOrdersInSequence(sequence, orders, numCustomers, numProducts);
    cout << "Open stacks: "
         << maximumOpenStacks(sequence, orders, numCustomers, numProducts)
         << endl;

    free(sequence);
    // End of debugging code
}

void dpSolve(int* orders) {
    // for each size of set generate the number of
}

int main(int argc, char** argv) {
    ifstream readFile;
    int numCustomers, numProducts;
    int* orders;

    char* input;
    if (argc > 1) {
        input = argv[1];
    } else {
        input = "input.txt";
    }
    cout << "Reading from " << input << endl;
    readFile.open(input);

    if (readFile.is_open()) {
        readFile >> numCustomers;
        readFile >> numProducts;
        orders = (int*) malloc(numCustomers * numProducts * sizeof(int));
        for (int i = 0; i < numCustomers; i++) {
            for (int j = 0; j < numProducts; j++) {
                int output;
                readFile >> output;
                orders[mIndex(i, j, numCustomers, numProducts)] = output;
            }
        }
        readFile.close();
    } else {
        cout << "Not able to open the input file." <<  endl;
    }
    cout << "numCustomers: " << numCustomers << endl
         << "numProducts: " << numProducts << endl;
    printOrders(orders, numCustomers, numProducts);
    clock_t start = clock();
    bruteForceSolve(orders, numCustomers, numProducts);
    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    cout << "Took " << seconds << " seconds" << endl;
    return 0;
}
