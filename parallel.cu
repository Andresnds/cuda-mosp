#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <time.h>

#define NUM_BLOCKS 65535

using namespace std;

__device__ __host__
int mIndex(int i, int j, int n, int m) {
    return i*m + j;
}

int64_t cache[100];
int64_t cached = 0;
int64_t factorial(int64_t n) {
    if (n < 2) return 1;
    if (n > cached) {
        cache[n] = n * factorial(n-1);
        cached = n;
        // cout << "Calculating factorial " << n <<": " << cache[n] << endl;
    }
    return cache[n];
}

void printOrders(int* orders, int numCustomers, int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, j, numCustomers, numProducts)] << " ";
        }
        cout << endl;
    }
}

void printSet(int set, int numProducts) {
    for (int i = 0; i < numProducts; i++) {
        cout << set % 2 << " ";
        set /= 2;
    }
    cout << endl;
}

void printOrdersInSequence(int* sequence,
                           int* orders,
                           int numCustomers,
                           int numProducts) {
    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            cout << orders[mIndex(i, sequence[j], numCustomers, numProducts)]
                 << " ";
        }
        cout << endl;
    }
}

// Calculates the maximum number of open stacks for a given producing sequence
__device__ __host__
int maximumOpenStacks(int* sequence,
                      int* orders,
                      int numCustomers,
                      int numProducts) {
    int* toDo = (int*) malloc(numCustomers * sizeof(int));
    int* done = (int*) malloc(numCustomers * sizeof(int));
    for (int customer = 0; customer < numCustomers; customer++) {
        done[customer] = 0;
        toDo[customer] = 0;
        for (int product = 0; product < numProducts; product++) {
            // suppose that orders has only 1's and 0's
            toDo[customer] += orders[mIndex(customer, product, numCustomers,
                                            numProducts)];
        }
    }

    int numOpenStacks = 0;
    for (int i = 0; i < numProducts; i++) {
        int product = sequence[i];
        for (int customer = 0; customer < numCustomers; customer++) {
            if (orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                toDo[customer]--;
                done[customer]++;
            }
        }
        int currentOpenStacks = 0;
        for (int customer = 0; customer < numCustomers; customer++) {
            if ((done[customer] > 0 && toDo[customer] > 0) ||
                orders[mIndex(customer,
                              product,
                              numCustomers,
                              numProducts)] > 0) {
                currentOpenStacks++;
            }
        }
        if (currentOpenStacks > numOpenStacks) {
            numOpenStacks = currentOpenStacks;
        }
    }

    free(toDo);
    free(done);

    return numOpenStacks;
}

__device__ __host__
void generateSequence(int* sequence, int k, int numProducts) {
    for (int i = 0; i < numProducts; i++) {
        sequence[i] = i;
    }
    for (int i = 0; i < numProducts; i++) {
        int temp = sequence[k % (i + 1)];
        sequence[k % (i + 1)] = sequence[i];
        sequence[i] = temp;
        k = k / (i + 1);
    }
}

__global__
void calculateMaximumOpenStacks(int* stackSizes,
                                int* orders,
                                int numCustomers,
                                int numProducts,
                                int step) {
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, step * NUM_BLOCKS + blockIdx.x, numProducts);
    stackSizes[blockIdx.x] = maximumOpenStacks(sequence,
                                               orders,
                                               numCustomers,
                                               numProducts);
    free(sequence);
}

void checkOk(hipError_t err) {
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " at " << __LINE__ << endl;
        exit(EXIT_FAILURE);

    }
}

void bruteForceSolve(int* orders,
                     int numCustomers,
                     int numProducts) {
    int* orders_d;
    int sizeOrders = numCustomers * numProducts * sizeof(int);
    checkOk(hipMalloc((void**) &orders_d, sizeOrders));
    checkOk(hipMemcpy(orders_d, orders, sizeOrders, hipMemcpyHostToDevice));

    int* stackSizes_d;
    int64_t numSequences = factorial(numProducts);
    int sizeStacksSizes = NUM_BLOCKS * sizeof(int);
    checkOk(hipMalloc((void**) &stackSizes_d, sizeStacksSizes));

    cout << "numSequences: " << numSequences << endl;

    int* stackSizes = (int*) malloc(sizeStacksSizes);
    int minStacks = numCustomers + 1;
    int bestK = -1;
    for (int i = 0; i < ceil(numSequences/NUM_BLOCKS); i++) {
        int numSequencesToProcess;
        if (numSequences - i * NUM_BLOCKS >= NUM_BLOCKS)
            numSequencesToProcess = NUM_BLOCKS;
        else
            numSequencesToProcess = numSequences - i * NUM_BLOCKS;

        // cout << "Step " << i << ". Calculating " << numSequencesToProcess
        //      << " More " << numSequences - i * NUM_BLOCKS << " to go." << endl;

        // Calculating maximum stack for each one of them
        calculateMaximumOpenStacks<<<numSequencesToProcess, 1>>>(stackSizes_d,
                                                                 orders_d,
                                                                 numCustomers,
                                                                 numProducts,
                                                                 i);

        checkOk(hipMemcpy(stackSizes,
                           stackSizes_d,
                           sizeStacksSizes,
                           hipMemcpyDeviceToHost));

        // Calculate the global minimum
        for (int j = 0; j < numSequencesToProcess; j++) {
            if (stackSizes[j] < minStacks) {
                minStacks = stackSizes[j];
                bestK = j + i * NUM_BLOCKS;
            }
        }
    }

    checkOk(hipFree(orders_d));
    checkOk(hipFree(stackSizes_d));

    free(stackSizes);

    // Debugging output

    cout << "minStacks: " << minStacks << endl;

    // Print sequence
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    generateSequence(sequence, bestK, numProducts);
    cout << "Best sequence:" << endl;
    for (int i = 0; i < numProducts; i++) {
        cout << sequence[i] << " ";
    }
    cout << endl;

    // See orders being produced
    printOrdersInSequence(sequence, orders, numCustomers, numProducts);
    cout << "Open stacks: "
         << maximumOpenStacks(sequence, orders, numCustomers, numProducts)
         << endl;

    free(sequence);
    // End of debugging code
}

int64_t combination(int n, int k) {
    return factorial(n)/factorial(k)/factorial(n-k);
}

__device__
bool contains(int set, int p) {
    for (int i = 0; i < p; i++) {
        set /= 2;
    }
    // cout << set << endl;
    return set % 2;
}

__device__ __host__
int remove(int set, int p) {
    int stack = 0;
    int offset = 1;
    for (int i = 0; i < 32; i++) {
        if (i != p)
            stack += (set % 2) * offset;
        set /= 2;
        offset *= 2;
    }
    return stack;
}

__device__
int a(int p,
      int set,
      int* orders,
      int numCustomers,
      int numProducts) {
    bool* before = (bool*) malloc(numCustomers * sizeof(bool));
    bool* after = (bool*) malloc(numCustomers * sizeof(bool));
    bool* now = (bool*) malloc(numCustomers * sizeof(bool));
    for (int i = 0; i < numCustomers; i++) {
        before[i] = false;
        after[i] = false;
        now[i] = false;
    }

    for (int i = 0; i < numCustomers; i++) {
        for (int j = 0; j < numProducts; j++) {
            if (j == p &&
                orders[mIndex(i, j, numCustomers, numProducts)] > 0) {
                now[i] = true;
            }
            if (contains(set, j) == true &&
                orders[mIndex(i, j, numCustomers, numProducts)] > 0) {
                after[i] = true;
            }
            if (contains(set, j) == false &&
                orders[mIndex(i, j, numCustomers, numProducts)] > 0) {
                before[i] = true;
            }
        }
    }

    // cout << "Testing after and before for set: " << set << " and product " << p << endl;
    // printSet(set, numProducts);

    int active_stacks = 0;
    for (int i = 0; i < numCustomers; i++) {
        if(now[i] || (before[i] && after[i])) {
            active_stacks++;
            // cout << i << " ";
        }
    }
    // cout << endl;

    free(now);
    free(after);
    free(before);

    // cout << active_stacks << " active" << endl;

    return active_stacks;
}

__global__
void computeStacks(int offset,
                   int* sets,
                   int* stacksResults,
                   int* bestP,
                   int* orders,
                   int numCustomers,
                   int numProducts) {
    int set = sets[offset + blockIdx.x];

    if (set == 0) {
        stacksResults[set] = 0;
        return;
    }

    int best = -1;
    int min_stacks = numCustomers * 10;
    for (int p = 0; p < numProducts; p++) {
        if (contains(set, p)) {
            int newSet = remove(set, p);
            int active = a(p, newSet, orders, numCustomers, numProducts);
            int after = stacksResults[newSet];
            int max = (active > after) ? active : after;
            if (max < min_stacks) {
                min_stacks = max;
                best = p;
            }
        }
    }

    stacksResults[set] = min_stacks;
    bestP[set] = best;
}

int countOnes(int n) {
    int ones = 0;
    while (n > 0) {
        ones += n % 2;
        n /= 2;
    }
    return ones;
}

void dpSolve(int* orders, int numCustomers, int numProducts) {
    int* orders_d;
    int sizeOrders = numCustomers * numProducts * sizeof(int);
    checkOk(hipMalloc((void**) &orders_d, sizeOrders));
    checkOk(hipMemcpy(orders_d, orders, sizeOrders, hipMemcpyHostToDevice));

    int** sorted_sets = (int**) malloc((numProducts + 1) * sizeof(int*));
    int* combinations = (int*) malloc((numProducts + 1) * sizeof(int*));
    for (int i = 0; i < (numProducts + 1); i++) {
        int numCombinations = combination(numProducts, i);
        sorted_sets[i] = (int*) malloc(numCombinations * sizeof(int));
        combinations[i] = 0;
    }

    for (int i = 0; i < pow(2, numProducts); i++) {
        int ones = countOnes(i);
        sorted_sets[ones][combinations[ones]] = i;
        combinations[ones]++;
    }

    int setsSize = pow(2, numProducts) * sizeof(int);
    int* sets = (int*) malloc(setsSize);
    int* sets_d;
    checkOk(hipMalloc((void**) &sets_d, setsSize));
    int i = 0;
    for (int j = 0; j < (numProducts + 1); j++) {
        for (int k = 0; k < combinations[j]; k++) {
            sets[i] = sorted_sets[j][k];
            i++;
        }
    }
    checkOk(hipMemcpy(sets_d, sets, setsSize, hipMemcpyHostToDevice));

    int stacksResultsSize = pow(2, numProducts) * sizeof(int);
    int* stacksResults = (int*) malloc(stacksResultsSize);
    int* stacksResults_d;
    checkOk(hipMalloc((void**) &stacksResults_d, stacksResultsSize));


    int bestPSize = pow(2, numProducts) * sizeof(int);
    int* bestP = (int*) malloc(bestPSize);
    int* bestP_d;
    checkOk(hipMalloc((void**) &bestP_d, bestPSize));

    int offset = 0;
    for (int setSize = 0;  setSize < numProducts + 1;  setSize++) {
        computeStacks<<<combinations[setSize], 1>>>(offset,
                                                    sets_d,
                                                    stacksResults_d,
                                                    bestP_d,
                                                    orders_d,
                                                    numCustomers,
                                                    numProducts);
        offset += combinations[setSize];
    }

    checkOk(hipMemcpy(stacksResults, stacksResults_d, stacksResultsSize, hipMemcpyDeviceToHost));

    checkOk(hipMemcpy(bestP, bestP_d, bestPSize, hipMemcpyDeviceToHost));

    // cout << "bestP" << endl;
    // for (int i = 0; i < pow(2, numProducts); i++) {
    //     cout << bestP[i] << " ";
    // }
    // cout << endl;

    // cout << "stacksResults" << endl;
    // for (int i = 0; i < pow(2, numProducts); i++) {
    //     cout << stacksResults[i] << " ";
    // }
    // cout << endl;

    cout << "Best sequence:" << endl;
    int set = pow(2, numProducts) - 1;
    int* sequence = (int*) malloc(numProducts * sizeof(int));
    for (int i = 0; i < numProducts; i++) {
        int best = bestP[set];
        set = remove(set, best);
        sequence[i] = best;
        cout << best << " ";
    }
    cout << endl;
    printOrdersInSequence(sequence, orders, numCustomers, numProducts);

    cout << "OpenStacks: " << stacksResults[(int) pow(2, numProducts) - 1] << endl;

    // Freeing memory

    checkOk(hipFree(bestP_d));
    checkOk(hipFree(stacksResults_d));
    checkOk(hipFree(sets_d));
    checkOk(hipFree(orders_d));

    free(bestP);
    free(stacksResults);
    free(sets);
    free(combinations);
    for (int i = 0; i < (numProducts + 1); i++) free(sorted_sets[i]);
    free(sorted_sets);

}

int main(int argc, char** argv) {
    ifstream readFile;
    int numCustomers, numProducts;
    int* orders;

    bool useBruteForce = false;
    if (argc < 1 || (strncmp(argv[1], "bf", 2) != 0 &&
                     strncmp(argv[1], "dp", 2) != 0)) {
        cout << "Specify if should use \"bf\" or \"dp\" as the first argument"
             << endl;
        exit(EXIT_FAILURE);
    } else {
        if (strncmp(argv[1], "bf", 2) == 0) {
            cout << "Solving by Brute Force..." << endl;
            useBruteForce = true;
        } else {
            cout << "Solving by Dynamic Programming..." << endl;
            useBruteForce = false;
        }
    }

    if (argc < 2) {
        cout << "Specify the input file as the second argument" << endl;
        exit(EXIT_FAILURE);
    }

    char* input;
    input = argv[2];
    cout << "Reading from " << input << endl;
    readFile.open(input);
    if (readFile.is_open()) {
        readFile >> numCustomers;
        readFile >> numProducts;
        orders = (int*) malloc(numCustomers * numProducts * sizeof(int));
        for (int i = 0; i < numCustomers; i++) {
            for (int j = 0; j < numProducts; j++) {
                int output;
                readFile >> output;
                orders[mIndex(i, j, numCustomers, numProducts)] = output;
            }
        }
        readFile.close();
    } else {
        cout << "Not able to open the input file." <<  endl;
    }
    cout << "numCustomers: " << numCustomers << endl
         << "numProducts: " << numProducts << endl;
    printOrders(orders, numCustomers, numProducts);

    clock_t start = clock();
    if (useBruteForce) {
        bruteForceSolve(orders, numCustomers, numProducts);
    }
    else {
        dpSolve(orders, numCustomers, numProducts);
    }
    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    cout << "Took " << seconds << " seconds" << endl;


    return 0;
}
