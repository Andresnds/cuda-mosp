#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

using namespace std;

void printDeviceProperties();

void checkOk(hipError_t err) {
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char** argv) {
    // cout << "ceil(13/2) = " << ceil(13/2) << endl;
    // cout << "ceil(12/2) = " << ceil(12/2) << endl;

    printDeviceProperties();
    return 0;
}

void printDeviceProperties() {

    int device = -1;
    checkOk(hipGetDevice(&device));
    cout << "Device: " << device << endl;

    hipDeviceProp_t prop;
    checkOk(hipGetDeviceProperties(&prop, device));
    cout << "Properties: "  << endl;

    // Those were added in CUDA 8.0
    // cout << "  hostNativeAtomicSupported: " << prop.hostNativeAtomicSupported << endl;
    // cout << "  concurrentManagedAccess: " << prop.concurrentManagedAccess << endl;
    // cout << "  pageableMemoryAccess: " << prop.pageableMemoryAccess << endl;
    // cout << "  singleToDoublePrecisionPerfRatio: " << prop.singleToDoublePrecisionPerfRatio << endl;


    cout << "  ECCEnabled: " << prop.ECCEnabled << endl;
    cout << "  asyncEngineCount: " << prop.asyncEngineCount << endl;
    cout << "  computeMode: " << prop.computeMode << endl;
    cout << "  concurrentKernels: " << prop.concurrentKernels << endl;
    cout << "  deviceOverlap: " << prop.deviceOverlap << endl;
    cout << "  globalL1CacheSupported: " << prop.globalL1CacheSupported << endl;
    cout << "  integrated: " << prop.integrated << endl;
    cout << "  isMultiGpuBoard: " << prop.isMultiGpuBoard << endl;
    cout << "  kernelExecTimeoutEnabled: " << prop.kernelExecTimeoutEnabled << endl;
    cout << "  l2CacheSize: " << prop.l2CacheSize << endl;
    cout << "  localL1CacheSupported: " << prop.localL1CacheSupported << endl;
    cout << "  major: " << prop.major << endl;
    cout << "  maxSurface1D: " << prop.maxSurface1D << endl;
    cout << "  maxSurface1DLayered[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxSurface1DLayered[i] << ", ";
    cout << endl;
    cout << "  maxSurface2D[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxSurface2D[i] << ", ";
    cout << endl;
    cout << "  maxSurface2DLayered[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxSurface2DLayered[i] << ", ";
    cout << endl;
    cout << "  maxSurface3D[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxSurface3D[i] << ", ";
    cout << endl;
    cout << "  maxSurfaceCubemap: " << prop.maxSurfaceCubemap << endl;
    cout << "  maxSurfaceCubemapLayered[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxSurfaceCubemapLayered[i] << ", ";
    cout << endl;
    cout << "  maxTexture1D: " << prop.maxTexture1D << endl;
    cout << "  maxTexture1DLayered[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxTexture1DLayered[i] << ", ";
    cout << endl;
    cout << "  maxTexture1DLinear: " << prop.maxTexture1DLinear << endl;
    cout << "  maxTexture1DMipmap: " << prop.maxTexture1DMipmap << endl;
    cout << "  maxTexture2D[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxTexture2D[i] << ", ";
    cout << endl;
    cout << "  maxTexture2DGather[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxTexture2DGather[i] << ", ";
    cout << endl;
    cout << "  maxTexture2DLayered[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxTexture2DLayered[i] << ", ";
    cout << endl;
    cout << "  maxTexture2DLinear[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxTexture2DLinear[i] << ", ";
    cout << endl;
    cout << "  maxTexture2DMipmap[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxTexture2DMipmap[i] << ", ";
    cout << endl;
    cout << "  maxTexture3D[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxTexture3D[i] << ", ";
    cout << endl;
    cout << "  maxTexture3DAlt[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxTexture3DAlt[i] << ", ";
    cout << endl;
    cout << "  maxTextureCubemap: " << prop.maxTextureCubemap << endl;
    cout << "  maxTextureCubemapLayered[2]: ";
    for (int i = 0; i < 2; i ++) cout << prop.maxTextureCubemapLayered[i] << ", ";
    cout << endl;
    cout << "  memPitch: " << prop.memPitch << endl;
    cout << "  memoryBusWidth: " << prop.memoryBusWidth << endl;
    cout << "  memoryClockRate: " << prop.memoryClockRate << endl;
    cout << "  minor: " << prop.minor << endl;
    cout << "  multiGpuBoardGroupID: " << prop.multiGpuBoardGroupID << endl;
    cout << "  multiProcessorCount: " << prop.multiProcessorCount << endl;
    cout << "  pciBusID: " << prop.pciBusID << endl;
    cout << "  pciDeviceID: " << prop.pciDeviceID << endl;
    cout << "  pciDomainID: " << prop.pciDomainID << endl;
    cout << "  regsPerBlock: " << prop.regsPerBlock << endl;
    cout << "  regsPerMultiprocessor: " << prop.regsPerMultiprocessor << endl;
    cout << "  sharedMemPerBlock: " << prop.sharedMemPerBlock << endl;
    cout << "  sharedMemPerMultiprocessor: " << prop.sharedMemPerMultiprocessor << endl;
    cout << "  streamPrioritiesSupported: " << prop.streamPrioritiesSupported << endl;
    cout << "  surfaceAlignment: " << prop.surfaceAlignment << endl;
    cout << "  tccDriver: " << prop.tccDriver << endl;
    cout << "  textureAlignment: " << prop.textureAlignment << endl;
    cout << "  texturePitchAlignment: " << prop.texturePitchAlignment << endl;

    cout << "---------------------------------" << endl;
    cout << "  name: " << prop.name << endl;
    cout << "  clockRate: " << prop.clockRate << endl;
    cout << "  canMapHostMemory: " << prop.canMapHostMemory << endl;
    cout << "  managedMemory: " << prop.managedMemory << endl;
    cout << "  maxGridSize[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxGridSize[i] << ", ";
    cout << endl;
    cout << "  maxThreadsDim[3]: ";
    for (int i = 0; i < 3; i ++) cout << prop.maxThreadsDim[i] << ", ";
    cout << endl;
    cout << "  maxThreadsPerBlock: " << prop.maxThreadsPerBlock << endl;
    cout << "  maxThreadsPerMultiProcessor: " << prop.maxThreadsPerMultiProcessor << endl;
    cout << "  totalConstMem: " << prop.totalConstMem << endl;
    cout << "  totalGlobalMem: " << prop.totalGlobalMem << endl;
    cout << "  unifiedAddressing: " << prop.unifiedAddressing << endl;
}